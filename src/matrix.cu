#include "hip/hip_runtime.h"
/**
 * \file	matrix_test.cu
 * \brief	
 * 
 * \author	Erlend Isachsen (erlend.isachsen@7sense.no)
 * 
*/

/** Related header include	*/
#include "matrix.cuh"

/** Standard library header includes	*/
#include <cstdint>
#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdexcept>
#include <sstream>

/** Project header includes	*/

namespace utils
{
	/** Variable declarations	*/
	const float h_kMinusOne	= -1.0F;
	const float h_kZero		= 0.0F;
	const float h_kUnity	= 1.0F;

	/** Funciton declarations	*/
	void test()
	{
		std::cout << "In test" << std::endl;
	}

	/** Classes, structs and enums	*/


	/** Funciton Implementations	 */
	Matrix::Matrix(const uint32_t n_cols, const uint32_t n_rows, float* mat) : 
		n_cols_	{ n_cols	},
		n_rows_	{ n_rows 	},
		mat_	{ mat		}
	{

	}

	Matrix::Matrix(const uint32_t n_cols, const uint32_t n_rows) :
		Matrix( n_cols, n_rows, (float*)malloc(sizeof(float) * n_cols * n_rows) )
	{

	}

	Matrix::Matrix(const Matrix &other) : 
		Matrix( other.n_cols_, other.n_rows_ )
	{
		memcpy(mat_, other.mat_, Size());
	}

	Matrix::Matrix(const DMatrix &d_mat) : 
		Matrix(d_mat.n_cols_, d_mat.n_rows_)
	{
		hipMemcpy(mat_, d_mat.mat_, Size(), hipMemcpyDeviceToHost);
	}

	Matrix::~Matrix()
	{
		if(mat_ != nullptr)
		{
			free(mat_);
			mat_ = nullptr;
		}
	}

	Matrix::Matrix(Matrix &&other) : 
		n_cols_	{ other.n_cols_ },
		n_rows_	{ other.n_rows_ },
		mat_	{ other.mat_ }
	{
		other.mat_ = nullptr;
		other.~Matrix();
	}

	const float *Matrix::Get() const
	{
		return mat_;
	}

	float Matrix::At(const uint32_t col_i, const uint32_t row_i) const
	{
		const size_t index = (size_t)n_rows_ * col_i + row_i;
		return mat_[index];
	}

	void Matrix::Set(const uint32_t col_i, const uint32_t row_i, const float val)
	{
		const size_t index = (size_t)n_rows_ * col_i + row_i;
		mat_[index] = val;
	}

	size_t Matrix::Size() const
	{
		return sizeof(float) * Length();
	}

	size_t Matrix::Length() const
	{
		return n_cols_ * n_rows_;
	}

	Matrix& Matrix::operator=(const Matrix& other)
	{
		if((this->n_cols_ != other.n_cols_) || (this->n_rows_ != other.n_rows_))
		{
			/** \todo throw eception */
		}

		memcpy( this->mat_, other.mat_, Size() );
		return *this;
	}

	Matrix& Matrix::operator=(const DMatrix& other)
	{
		if((this->n_cols_ != other.n_cols_) || (this->n_rows_ != other.n_rows_))
		{
			/** \todo throw eception */
		}

		hipMemcpy(this->mat_, other.mat_, Size(), hipMemcpyDeviceToHost);
		return *this;
	}

	bool Matrix::operator==(const Matrix& other) const
	{
		bool equal = true;
		if(this->n_cols_ != other.n_cols_)
			equal = false;
		else if(this->n_rows_ != other.n_rows_)
			equal = false;
		
		for(size_t col_i = 0; (col_i < n_cols_) && (equal); col_i++)
		{
			for(size_t row_i = 0; (row_i < n_rows_) && (equal); row_i++)
			{
				equal = fabsf(this->At(col_i, row_i) - other.At(col_i, row_i)) < FLT_EPSILON;
				equal = equal || (isnan(this->At(col_i, row_i) ) && other.At(col_i, row_i));
			}
		}

		return equal;
	}

	std::ostream& operator<<(std::ostream& os, const Matrix& m)
	{
		os << std::fixed << std::setprecision(2);
		os << "{";
		for(uint32_t row_i = 0; row_i < m.n_rows_; row_i++)
		{
			if(row_i != 0)
				os << "," << std::endl << "{ ";
			else
				os << "{";

			for(uint32_t col_i = 0; col_i < m.n_cols_; col_i++)
			{
				if(col_i != 0)
					os << ",";
				os << std::setw(9) << std::scientific << m.At(col_i, row_i);
			}
			os << " }";
		}
		os << "}";

		return os;
	}

	DMatrix::DMatrix(const uint32_t n_cols, const uint32_t n_rows) : 
		Matrix(n_cols, n_rows, nullptr),
		cublas_handle_{}
	{
		// Allocate the matrix buffer on the device.
		const hipError_t cuda_err = hipMalloc(&mat_, Size());
		if (hipSuccess != cuda_err)
		{
			std::stringstream err_msg;
			err_msg << "Runtime error in [" << __PRETTY_FUNCTION__ << "]. "
					<< "The function 'hipMalloc' returned with error code [" << cuda_err << "].";
			throw(std::runtime_error(err_msg.str()));
		}
		
		// Create the cublas handle structure
		const hipblasStatus_t cublas_err = hipblasCreate(&cublas_handle_);
		if (HIPBLAS_STATUS_SUCCESS != cublas_err)
		{
			std::stringstream err_msg;
			err_msg << "Runtime error in [" << __PRETTY_FUNCTION__ << "]. "
					<< "The function 'hipblasCreate' returned with error code [" << cublas_err << "].";
			throw( std::runtime_error(err_msg.str()) );
		}
	}

	DMatrix::DMatrix(const DMatrix &other) : 
		DMatrix(other.n_cols_, other.n_rows_)
	{
		const hipError_t cuda_err = hipMemcpy(mat_, other.mat_, Size(), hipMemcpyDeviceToDevice);
		if (hipSuccess != cuda_err)
		{
			std::stringstream err_msg;
			err_msg << "Runtime error in [" << __PRETTY_FUNCTION__ << "] while attempting to copy data from device to device. "
					<< "The function 'hipMemcpy' returned with error code [" << cuda_err << "].";
			throw(std::runtime_error(err_msg.str()));
		}
	}

	DMatrix::DMatrix(DMatrix &&other) : 
		Matrix(other.n_cols_, other.n_rows_, other.mat_),
		cublas_handle_{ other.cublas_handle_ }
	{
		other.mat_ = nullptr;
		other.cublas_handle_ = nullptr;
	}

	DMatrix::DMatrix(const Matrix &h_mat) : 
		DMatrix(h_mat.n_cols_, h_mat.n_rows_)
	{
		const hipError_t cuda_err = hipMemcpy(mat_, h_mat.Get(), Size(), hipMemcpyHostToDevice);
		if (hipSuccess != cuda_err)
		{
			std::stringstream err_msg;
			err_msg << "Runtime error in [" << __PRETTY_FUNCTION__ << "] while attempting to copy data from host to device. "
					<< "The function 'hipMemcpy' returned with error code [" << cuda_err << "].";
			throw(std::runtime_error(err_msg.str()));
		}
	}

	DMatrix::DMatrix(const std::vector<std::vector<float>> &mat) :
		DMatrix( mat.front().size(), mat.size() )
	{
		for (size_t row_i = 0; row_i < mat.size(); row_i++)
		{
			if (mat.at(row_i).size() != n_cols_)
			{
				std::stringstream err_msg;
				err_msg << "Dimension error in [" << __PRETTY_FUNCTION__ << "] while constructing matrix from vector. "
						<< "All rows must be of equal size.";
				throw(std::runtime_error(err_msg.str()));
			}
		}

		for (size_t row_i = 0; row_i < n_rows_; row_i++)
		{
			for (size_t col_i = 0; col_i < n_cols_; col_i++)
			{
				Set(col_i, row_i, mat.at(row_i).at(col_i));
			}
		}
	}

	DMatrix::~DMatrix()
	{
		if(mat_ != nullptr)
		{
			hipFree(mat_);
			mat_ = nullptr;
		}
		if(cublas_handle_ != nullptr)
		{
			hipblasDestroy(cublas_handle_);
			cublas_handle_ = nullptr;
		}
	}

	float DMatrix::At(const uint32_t col_i, uint32_t row_i) const
	{
		float h_val;
		const hipError_t cuda_err = hipMemcpy(&h_val, &(mat_[n_rows_ * col_i + row_i]), sizeof(float), hipMemcpyDeviceToHost);
		if (hipSuccess != cuda_err)
		{
			std::stringstream err_msg;
			err_msg << "Runtime error in [" << __PRETTY_FUNCTION__ << "] while attempting to copy data from device to host. "
					<< "The function 'hipMemcpy' returned with error code [" << cuda_err << "].";
			throw(std::runtime_error(err_msg.str()));
		}
		return h_val;
	}

	void DMatrix::Set(const uint32_t col_i, const uint32_t row_i, const float val)
	{
		const hipError_t cuda_err = hipMemcpy(&(mat_[n_rows_ * col_i + row_i]), &val, sizeof(float), hipMemcpyHostToDevice);
		if (hipSuccess != cuda_err)
		{
			std::stringstream err_msg;
			err_msg << "Runtime error in [" << __PRETTY_FUNCTION__ << "] while attempting to copy data from host to device. "
					<< "The function 'hipMemcpy' returned with error code [" << cuda_err << "].";
			throw(std::runtime_error(err_msg.str()));
		}
	}

	DMatrix& DMatrix::operator=(const DMatrix& other)
	{
		if((this->n_cols_ != other.n_cols_) || (this->n_rows_ != other.n_rows_))
		{
			std::stringstream err_msg;
			err_msg << "Runtime error in [" << __PRETTY_FUNCTION__ << "]. Incompatible sizes. The two matrices must have equal dimensions.";
			throw(std::runtime_error(err_msg.str()));
		}
		hipMemcpy(mat_, other.Get(), Size(), hipMemcpyDeviceToDevice);
		return *this;
	}

	DMatrix& DMatrix::operator=(const Matrix& other)
	{
		if((this->n_cols_ != other.n_cols_) || (this->n_rows_ != other.n_rows_))
		{
			std::stringstream err_msg;
			err_msg << "Runtime error in [" << __PRETTY_FUNCTION__ << "]."
					<< "The two matrices have incompatible sizes.The two matrices must have equal dimensions.";
			throw(std::runtime_error(err_msg.str()));
		}
		hipMemcpy(mat_, other.Get(), Size(), hipMemcpyHostToDevice);
		return *this;
	}

	DMatrix& DMatrix::operator=(const std::vector<std::vector<float>> &matrix)
	{
		*this = DMatrix(matrix);
		return *this;
	}

	DMatrix &DMatrix::operator-()
	{
		const hipblasStatus_t cublas_err = hipblasSscal(cublas_handle_, Length(), &h_kMinusOne, mat_, 1);
		if (HIPBLAS_STATUS_SUCCESS != cublas_err)
		{
			std::stringstream err_msg;
			err_msg << "Runtime error in [" << __PRETTY_FUNCTION__ << "]. "
					<< "The function 'hipblasSscal' returned with error code [" << cublas_err << "].";
			throw(std::runtime_error(err_msg.str()));
		}
		return *this;
	}


	DMatrix& DMatrix::operator += (const DMatrix& rhs)
	{
		const hipblasStatus_t cublas_err = hipblasSaxpy(cublas_handle_, Length(), &h_kUnity, rhs.mat_, 1, this->mat_, 1);
		if (HIPBLAS_STATUS_SUCCESS != cublas_err)
		{
			std::stringstream err_msg;
			err_msg << "Runtime error in [" << __PRETTY_FUNCTION__ << "]. "
				<< "The function 'hipblasSaxpy' returned with error code [" << cublas_err << "].";
			throw(std::runtime_error(err_msg.str()));
		}
		return *this;
	}
	
	DMatrix& DMatrix::operator *= (const float d_rhs)
	{
		const hipblasStatus_t cublas_err = hipblasSscal(cublas_handle_, Size(), &d_rhs, mat_, 1);
		if (HIPBLAS_STATUS_SUCCESS != cublas_err)
		{
			std::stringstream err_msg;
			err_msg << "Runtime error in [" << __PRETTY_FUNCTION__ << "]. "
				<< "The function 'hipblasSscal' returned with error code [" << cublas_err << "].";
			throw(std::runtime_error(err_msg.str()));
		}
		return *this;
	}

	DMatrix& DMatrix::operator *= (const float* d_rhs)
	{
		const hipblasStatus_t cublas_err = hipblasSscal(cublas_handle_, Size(), d_rhs, mat_, 1);
		if (HIPBLAS_STATUS_SUCCESS != cublas_err)
		{
			std::stringstream err_msg;
			err_msg << "Runtime error in [" << __PRETTY_FUNCTION__ << "]. "
				<< "The function 'hipblasSscal' returned with error code [" << cublas_err << "].";
			throw(std::runtime_error(err_msg.str()));
		}
		return *this;
	}
		
	DMatrix operator+(DMatrix lhs, const DMatrix& rhs)
	{
		lhs += rhs;
		return lhs;
	}

	DMatrix operator*(const DMatrix &lhs, const DMatrix& rhs)
	{
		if(lhs.n_cols_ != rhs.n_rows_)
		{
			std::stringstream err_msg;
			err_msg << "Runtime error in [" << __PRETTY_FUNCTION__ << "]. The two matrices have incompatible size. "
				<< "The number of columns in left hand side must equal the number of rows in right hand side. "
				<< "The matrices are: " << std::endl << lhs << "," << std::endl << rhs;
			throw(std::runtime_error(err_msg.str()));
		}

		DMatrix result(rhs.n_cols_, lhs.n_rows_);

		const hipblasStatus_t cublas_err = hipblasSgemm(
			lhs.cublas_handle_, HIPBLAS_OP_N, HIPBLAS_OP_N,
			result.n_rows_, result.n_cols_, lhs.n_cols_,
			&h_kUnity, lhs.mat_, lhs.n_rows_, rhs.mat_, rhs.n_rows_, &h_kZero, result.mat_, result.n_rows_
		);

		if (HIPBLAS_STATUS_SUCCESS != cublas_err)
		{
			std::stringstream err_msg;
			err_msg << "Runtime error in [" << __PRETTY_FUNCTION__ << "]. "
				<< "The function 'hipblasSgemm' returned with error code [" << cublas_err << "].";
			throw(std::runtime_error(err_msg.str()));
		}

		return result;
	}


} // Namespace ai


/*
 *	--- End of file ---
 */